#include "hip/hip_runtime.h"
#include <stdlib.h>
#include <string.h>
#include <stdio.h>
#include <hiprand.h>
#include <thrust/host_vector.h>
#define DEBUG
__device__ int devData;

__global__ void helloFromGPU(){
        //printf("THREAD ID: %d\n", threadIdx.x);
        //printf("BLOCK ID: %d\n", blockDim.x * blockIdx.x + threadIdx.x);
        //printf("THREADS ON BLOCK: %d\n", blockDim.x);
        //printf("BLOCKS ON GRID:%d\n", gridDim.x);
        atomicAdd(&devData, 1);
        printf("devData @ thread id %d: %d\n", blockDim.x * blockIdx.x + threadIdx.x, devData);
}

int main(void)
{
    int f = 0;
    hipMemcpyToSymbol(HIP_SYMBOL(devData), &f, sizeof(int));

    hipDeviceProp_t deviceProp;
    hipGetDeviceProperties(&deviceProp, 0);
    short MPC = deviceProp.multiProcessorCount;
    #ifdef DEBUG
    printf("MULTI-PROCESSOR COUNT TO BE USED IN TESTING: %d", MPC);
    #endif

    helloFromGPU <<<32, 32>>>();


    hipDeviceSynchronize();
    int outputData;
    hipMemcpyFromSymbol(&outputData, HIP_SYMBOL(devData), sizeof(int));
    if(outputData == 1024){
        printf("SUCCESSFUL RUN!");
    }else{
        printf("OUTPUT DATA IS: %d!\n", outputData);
    }

    hipDeviceReset();
    return 0;
}
